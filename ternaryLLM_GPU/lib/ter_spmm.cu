#include "ter_spmm.cuh"

/**
 * wrapper function for cusparse spmm api
 */
__host__ void Ternary_SpMM::ter_spmm_cusparse_spmm(TerSparseDataWrap<float>& spmm, SpMMStat& stat) {
    // CUDA_CALL_CHECK(hipSetDevice(0));
    // CUDA_CALL_CHECK(hipDeviceReset()); /* reset is needed to count overhead */
    auto fn_s = std::chrono::high_resolution_clock::now();
    /* csc device pointers */
    int* dev_w1_row_indices = 0;
    int* dev_w1_col_offset = 0;
    int8_t* dev_w1_values = 0;

    /* allocate device memory */
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_w1_row_indices), spmm.w1_cnt_nnz*sizeof(int)));
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_w1_col_offset), (spmm.columns+1)*sizeof(int)));
    CUDA_CALL_CHECK(hipMalloc((void**)(&dev_w1_values), spmm.w1_cnt_nnz*sizeof(float)));

    CUDA_CALL_CHECK(hipMalloc((void**)(&spmm.dev_x), spmm.size_x*sizeof(float)));
    CUDA_CALL_CHECK(hipMalloc((void**)(&spmm.dev_res), spmm.size_res*sizeof(float)));

    stat.fn_mem_use[stat.curr_config][spmm.sparsity] = (
        spmm.w1_cnt_nnz*sizeof(int) + 
        (spmm.columns+1)*sizeof(int) + 
        spmm.w1_cnt_nnz*sizeof(float) +
        spmm.size_x*sizeof(float) + spmm.size_res*sizeof(float)
    );

    /* copy csc data */
    CUDA_CALL_CHECK(hipMemcpy((void*)dev_w1_row_indices, (void*)spmm.w1_row_indice, spmm.w1_cnt_nnz*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CALL_CHECK(hipMemcpy((void*)dev_w1_col_offset, (void*)spmm.w1_col_offset, (spmm.columns+1)*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CALL_CHECK(hipMemcpy((void*)dev_w1_values, (void*)spmm.w1_values, spmm.w1_cnt_nnz*sizeof(float), hipMemcpyHostToDevice));

    CUDA_CALL_CHECK(hipMemcpy((void*)spmm.dev_x, (void*)spmm.host_x.data(), spmm.size_x*sizeof(float), hipMemcpyHostToDevice));

    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matW1;
    hipsparseDnMatDescr_t matX, matRes;

    CUSPARSE_CALL_CHECK(hipsparseCreate(&handle))

    // Create sparse matrix W1 in CSC format
    CUSPARSE_CALL_CHECK(hipsparseCreateCsc(&matW1, spmm.inners, spmm.columns, 
                        spmm.w1_cnt_nnz, dev_w1_col_offset, dev_w1_row_indices, dev_w1_values, 
                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    CUSPARSE_CALL_CHECK(hipsparseCreateDnMat(&matX, spmm.rows, spmm.inners, spmm.rows, spmm.dev_x, HIP_R_32F, HIPSPARSE_ORDER_COL));

    CUSPARSE_CALL_CHECK(hipsparseCreateDnMat(&matRes, spmm.columns, spmm.rows, spmm.columns, spmm.dev_res, HIP_R_32F, HIPSPARSE_ORDER_COL));
    
    // buffer
    void*                dBuffer    = 0;
    size_t               bufferSize = 0;
    float alpha           = 1.0f;
    float beta            = 0.0f;
    CUSPARSE_CALL_CHECK(hipsparseSpMM_bufferSize(handle,
                            HIPSPARSE_OPERATION_TRANSPOSE,
                            HIPSPARSE_OPERATION_TRANSPOSE,
                            &alpha, matW1, matX, &beta, matRes, HIP_R_32F,
                            HIPSPARSE_SPMM_CSR_ALG1, &bufferSize));
    CUDA_CALL_CHECK(hipMalloc(&dBuffer, bufferSize));
    
    // 
    hipEvent_t start, stop;
    float kn_span = 0;  // ms
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CUSPARSE_CALL_CHECK(hipsparseSpMM(handle,
                HIPSPARSE_OPERATION_TRANSPOSE,
                HIPSPARSE_OPERATION_TRANSPOSE,
                &alpha, matW1, matX, &beta, matRes, HIP_R_32F,
                HIPSPARSE_SPMM_CSR_ALG1, dBuffer));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kn_span, start, stop);
    stat.kn_spans[stat.curr_config][spmm.sparsity].push_back(kn_span);

    // destroy matrix/vector descriptors
    CUSPARSE_CALL_CHECK(hipsparseDestroySpMat(matW1));
    CUSPARSE_CALL_CHECK(hipsparseDestroyDnMat(matX));
    CUSPARSE_CALL_CHECK(hipsparseDestroyDnMat(matRes));
    CUSPARSE_CALL_CHECK(hipsparseDestroy(handle));


    CUDA_CALL_CHECK(hipMemcpy((void*)spmm.host_res.data(), (void*)spmm.dev_res, spmm.size_res*sizeof(float), hipMemcpyDeviceToHost));

    /* free */
    CUDA_CALL_CHECK(hipFree(dev_w1_col_offset));
    CUDA_CALL_CHECK(hipFree(dev_w1_row_indices));
    CUDA_CALL_CHECK(hipFree(dev_w1_values));
    CUDA_CALL_CHECK(hipFree(dBuffer));
    CUDA_CALL_CHECK(hipFree(spmm.dev_x));
    CUDA_CALL_CHECK(hipFree(spmm.dev_w1));
    CUDA_CALL_CHECK(hipFree(spmm.dev_res));

    auto fn_e = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> fn_span = fn_e - fn_s;
    stat.fn_spans[stat.curr_config][spmm.sparsity].push_back(fn_span.count());

    INFO(spmm.sparsity << " Memory use: " << stat.fn_mem_use[stat.curr_config][spmm.sparsity]/1024 << "KB"<< " Kernel span: " << kn_span << "ms" << " Function span: " << fn_span.count() << "ms");
    return;
}

/**
 * wrapper function for cublas gemm api
 */
__host__ void Ternary_SpMM::ter_spmm_cublas_spmm(TerSparseDataWrap<float>& spmm, SpMMStat& stat) {
    // CUDA_CALL_CHECK(hipSetDevice(0));
    // CUDA_CALL_CHECK(hipDeviceReset()); /* reset is needed to count overhead */

    hipblasHandle_t cublasH = NULL;
    // hipStream_t stream = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    float* dev_w1_float = 0;
    auto fn_s = std::chrono::high_resolution_clock::now();  // function runtime start
    auto prekn_s = std::chrono::high_resolution_clock::now();   // pre-kernel runtime start

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CALL_CHECK(hipblasCreate(&cublasH));

    // CUDA_CALL_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    // CUBLAS_CALL_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CALL_CHECK(hipMalloc(reinterpret_cast<void **>(&spmm.dev_x), sizeof(float) * spmm.size_x));
    CUDA_CALL_CHECK(hipMalloc(reinterpret_cast<void **>(&dev_w1_float), sizeof(float) * spmm.size_w1));
    CUDA_CALL_CHECK(hipMalloc(reinterpret_cast<void **>(&spmm.dev_res), sizeof(float) * spmm.size_res));
    stat.fn_mem_use[stat.curr_config][spmm.sparsity] = (
        sizeof(float) * spmm.size_x + 
        sizeof(float) * spmm.size_w1 + 
        sizeof(float) * spmm.size_res
    );

    CUDA_CALL_CHECK(hipMemcpy(spmm.dev_x, spmm.host_x.data(), sizeof(float) * spmm.size_x, hipMemcpyHostToDevice));
    CUDA_CALL_CHECK(hipMemcpy(dev_w1_float, spmm.host_w1_xtype.data(), sizeof(float) * spmm.size_w1, hipMemcpyHostToDevice));

    auto prekn_e = std::chrono::high_resolution_clock::now();   // pre-kernel runtime end
    std::chrono::duration<double, std::milli> prekn_span = prekn_e - prekn_s; // data prepare duration before API call


    /* step 3: compute */
    const float alpha = 1.0;
    const float beta = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipblasSgemm(cublasH, transa, transb, spmm.rows, spmm.columns, spmm.inners, &alpha, 
                spmm.dev_x, spmm.rows, 
                dev_w1_float, spmm.inners, &beta, 
                spmm.dev_res, spmm.rows);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float kn_span = 0;
    hipEventElapsedTime(&kn_span, start, stop);
    stat.kn_spans[stat.curr_config][spmm.sparsity].push_back(kn_span);

    auto postkn_s = std::chrono::high_resolution_clock::now();  // post-kernel start

    /* step 4: copy data to host */
    CUDA_CALL_CHECK(hipMemcpy(spmm.host_res.data(), spmm.dev_res, sizeof(float) * spmm.size_res, hipMemcpyDeviceToHost));


    /* free resources */
    CUDA_CALL_CHECK(hipFree(spmm.dev_res));
    CUDA_CALL_CHECK(hipFree(spmm.dev_x));
    CUDA_CALL_CHECK(hipFree(dev_w1_float));

    CUBLAS_CALL_CHECK(hipblasDestroy(cublasH));

    // CUDA_CALL_CHECK(hipStreamDestroy(stream));

    // CUDA_CALL_CHECK(hipDeviceReset());
    
    auto fn_e = std::chrono::high_resolution_clock::now();  // function runtime end
    std::chrono::duration<double, std::milli> postkn_span = fn_e - postkn_s;    // post-kernel duration
    std::chrono::duration<double, std::milli> fn_span = fn_e - fn_s;            // function duration
    stat.fn_spans[stat.curr_config][spmm.sparsity].push_back(fn_span.count());
    
    INFO(spmm.sparsity << " Memory: " << stat.fn_mem_use[stat.curr_config][spmm.sparsity]/1024 << "KB"<< " Kernel: " << kn_span << "ms"<< " Pre: " << prekn_span.count() << "ms"<< " Post: " << postkn_span.count() << "ms"<< " Function: " << fn_span.count() << "ms");
}